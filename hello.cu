
#include <hip/hip_runtime.h>
#include <iostream>

// A CUDA kernel (function executed on the GPU)
__global__ void say_hello() {
    printf("Hello, World from the GPU!\n");
}

int main() {
    // Call the kernel with one block containing one thread
    say_hello<<<1,1>>>(); 

    // Ensure all GPU tasks complete before exiting
    hipDeviceSynchronize(); 
    return 0; 
}